#include "hip/hip_runtime.h"
#include "bdd_cuda_base.h"
#include <thrust/sort.h>
#include <thrust/for_each.h>
#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>

namespace LPMP {
    // copied from: https://github.com/treecode/Bonsai/blob/8904dd3ebf395ccaaf0eacef38933002b49fc3ba/runtime/profiling/derived_atomic_functions.h#L186
    __device__ __forceinline__ float atomicMin(float *address, float val) //TODO: Check!
    {
        int ret = __float_as_int(*address);
        while(val < __int_as_float(ret))
        {
            int old = ret;
            if((ret = atomicCAS((int *)address, old, __float_as_int(val))) == old)
                break;
        }
        return __int_as_float(ret);
    }

    struct assign_new_indices_func {
        int* new_indices;
        __host__ __device__ void operator()(int& idx)
        {
            if(idx >= 0) // non-terminal nodes.
                idx = new_indices[idx];
        }
    };

    struct not_equal_to
    {
        int* values;
        int val_to_search;
        __host__ __device__
        bool operator()(const int i) const
        {
            return values[i] != val_to_search;
        }
    };

    template<typename T>
    void print_vector(const thrust::device_vector<T>& v, const char* name, const int num = 0)
    {
        std::cout<<name<<": ";
        if (num == 0)
            thrust::copy(v.begin(), v.end(), std::ostream_iterator<T>(std::cout, " "));
        else
        {
            int size = std::distance(v.begin(), v.end());
            thrust::copy(v.begin(), v.begin() + std::min(size, num), std::ostream_iterator<T>(std::cout, " "));
        }
        std::cout<<"\n";
    }

    bdd_cuda_base::bdd_cuda_base(BDD::bdd_collection& bdd_col)
    {
        std::vector<int> primal_variable_index; //TODO: Possibly store in compressed format?
        std::vector<int> lo_bdd_node_index;
        std::vector<int> hi_bdd_node_index;
        // Store hop distance from root node, so that all nodes with same hop distance can be processed in parallel:
        std::vector<int> bdd_hop_dist_root;
        std::vector<int> bdd_indices; // which bdd index does the bdd node belong to.
        std::vector<int> num_vars_per_bdd;

        std::unordered_map<size_t, int> primal_var_count;
        //TODO: Iterate over BDDs in sorted order w.r.t number of nodes.
        int storage_offset = 0;
        num_dual_variables_ = 0;
        for(size_t bdd_idx=0; bdd_idx < bdd_col.nr_bdds(); ++bdd_idx)
        {
            assert(bdd_col.is_qbdd(bdd_idx));
            assert(bdd_col.is_reordered(bdd_idx));
            int cur_hop_dist = 0;
            size_t prev_var = bdd_col(bdd_idx, storage_offset).index;
            for(size_t bdd_node_idx=0; bdd_node_idx < bdd_col.nr_bdd_nodes(bdd_idx); ++bdd_node_idx)
            {
                const auto cur_instr = bdd_col(bdd_idx, bdd_node_idx + storage_offset);
                const size_t var = cur_instr.index;
                if(prev_var != var)
                {
                    assert(prev_var < var);
                    prev_var = var;
                    if(!cur_instr.is_topsink())
                        cur_hop_dist++; // both terminal nodes can have same hop distance.
                }
                if(!cur_instr.is_terminal())
                {
                    assert(bdd_node_idx < bdd_col.nr_bdd_nodes(bdd_idx) - 2); // only last two nodes can be terminal nodes. 
                    primal_variable_index.push_back(var);
                    lo_bdd_node_index.push_back(cur_instr.lo);
                    hi_bdd_node_index.push_back(cur_instr.hi);
                }
                else if(cur_instr.is_topsink())
                {
                    primal_variable_index.push_back(-1);
                    lo_bdd_node_index.push_back(-1);
                    hi_bdd_node_index.push_back(-1);
                    assert(bdd_node_idx >= bdd_col.nr_bdd_nodes(bdd_idx) - 2);
                }
                else
                {
                    assert(cur_instr.is_botsink());
                    primal_variable_index.push_back(-2);
                    lo_bdd_node_index.push_back(-2);
                    hi_bdd_node_index.push_back(-2);
                    assert(bdd_node_idx >= bdd_col.nr_bdd_nodes(bdd_idx) - 2);
                }
                bdd_hop_dist_root.push_back(cur_hop_dist);
                bdd_indices.push_back(bdd_idx);
            }
            const std::vector<size_t> cur_bdd_variables = bdd_col.variables(bdd_idx);
            for (const auto& var : cur_bdd_variables) {
                auto it = primal_var_count.find(var);
                if(it != primal_var_count.end())
                    it->second++;
                else
                    primal_var_count[var] = 1;
            }

            num_vars_per_bdd.push_back(cur_bdd_variables.size());
            num_dual_variables_ += cur_bdd_variables.size();
            storage_offset += bdd_col.nr_bdd_nodes(bdd_idx);
        }
        // copy to GPU
        thrust::device_vector<int> primal_variable_index_temp(primal_variable_index.begin(), primal_variable_index.end());
        thrust::device_vector<int> bdd_index_temp(bdd_indices.begin(), bdd_indices.end());
        thrust::device_vector<int> lo_bdd_node_index_temp(lo_bdd_node_index.begin(), lo_bdd_node_index.end());
        thrust::device_vector<int> hi_bdd_node_index_temp(hi_bdd_node_index.begin(), hi_bdd_node_index.end());
        thrust::device_vector<int> bdd_hop_dist(bdd_hop_dist_root.begin(), bdd_hop_dist_root.end());
        cost_from_root_ = thrust::device_vector<float>(lo_bdd_node_index.size(), HIP_INF_F);
        cost_from_terminal_ = thrust::device_vector<float>(lo_bdd_node_index.size(), HIP_INF_F);
        hi_cost_ = thrust::device_vector<float>(lo_bdd_node_index.size(), HIP_INF_F);
        hi_path_cost_ = thrust::device_vector<float>(lo_bdd_node_index.size(), HIP_INF_F);
        lo_path_cost_ = thrust::device_vector<float>(lo_bdd_node_index.size(), HIP_INF_F);
        num_vars_per_bdd_ = thrust::device_vector<int>(num_vars_per_bdd.begin(), num_vars_per_bdd.end());

        // At this point all nodes of a BDD are contiguous in memory. Now we convert this so that nodes with same
        // hop distances become contiguous.

        // Determine ordering:
        thrust::device_vector<int> sorting_order(lo_bdd_node_index.size());
        thrust::sequence(sorting_order.begin(), sorting_order.end());
        thrust::sort_by_key(bdd_hop_dist.begin(), bdd_hop_dist.end(), sorting_order.begin());

        // Sort BDD nodes:
        primal_variable_index_ = thrust::device_vector<int>(primal_variable_index_temp.size());
        bdd_index_ = thrust::device_vector<int>(bdd_index_temp.size());
        lo_bdd_node_index_ = thrust::device_vector<int>(lo_bdd_node_index_temp.size());
        hi_bdd_node_index_ = thrust::device_vector<int>(hi_bdd_node_index_temp.size());
        thrust::gather(sorting_order.begin(), sorting_order.end(), primal_variable_index_temp.begin(), primal_variable_index_.begin());
        thrust::gather(sorting_order.begin(), sorting_order.end(), bdd_index_temp.begin(), bdd_index_.begin());
        thrust::gather(sorting_order.begin(), sorting_order.end(), lo_bdd_node_index_temp.begin(), lo_bdd_node_index_.begin());
        thrust::gather(sorting_order.begin(), sorting_order.end(), hi_bdd_node_index_temp.begin(), hi_bdd_node_index_.begin());
        
        // Since the ordering is changed so lo, hi indices also need to be updated:
        thrust::device_vector<int> new_indices(sorting_order.size());
        thrust::scatter(thrust::make_counting_iterator<int>(0), thrust::make_counting_iterator<int>(0) + sorting_order.size(), 
                        sorting_order.begin(), new_indices.begin());
        assign_new_indices_func func({thrust::raw_pointer_cast(new_indices.data())});
        thrust::for_each(lo_bdd_node_index_.begin(), lo_bdd_node_index_.end(), func);
        thrust::for_each(hi_bdd_node_index_.begin(), hi_bdd_node_index_.end(), func);

        // Count number of BDD nodes per hop distance:
        cum_nr_bdd_nodes_per_hop_dist_ = thrust::device_vector<int>(lo_bdd_node_index.size());
        auto last_red = thrust::reduce_by_key(bdd_hop_dist.begin(), bdd_hop_dist.end(), thrust::make_constant_iterator<int>(1), 
                                                thrust::make_discard_iterator(), 
                                                cum_nr_bdd_nodes_per_hop_dist_.begin());
        cum_nr_bdd_nodes_per_hop_dist_.resize(thrust::distance(cum_nr_bdd_nodes_per_hop_dist_.begin(), last_red.second));
        assert(cum_nr_bdd_nodes_per_hop_dist_[0] == bdd_col.nr_bdds()); // root nodes are 0 distance away and each BDD has exactly one root node.

        // Convert to cumulative:
        thrust::inclusive_scan(cum_nr_bdd_nodes_per_hop_dist_.begin(), cum_nr_bdd_nodes_per_hop_dist_.end(), cum_nr_bdd_nodes_per_hop_dist_.begin());

        nr_vars_ = *thrust::max_element(primal_variable_index_.begin(), primal_variable_index_.end()) + 1;
        nr_bdds_ = bdd_col.nr_bdds();
        nr_bdd_nodes_ = lo_bdd_node_index.size();

        // Populate variable counts:
        assert(primal_var_count.size() == nr_vars_);

        std::vector<int> primal_variable_counts(nr_vars_);
        for (const auto& [var, val] : primal_var_count) {
            primal_variable_counts[var] = val;
        }
        primal_variable_counts_ = thrust::device_vector<int>(primal_variable_counts.begin(), primal_variable_counts.end());

        // Set indices of BDD nodes which are root, top, bot sinks.
        root_indices_ = thrust::device_vector<int>(primal_variable_index_.size());
        thrust::sequence(root_indices_.begin(), root_indices_.end());

        auto last_root = thrust::remove_if(root_indices_.begin(), root_indices_.end(),
                                            not_equal_to({thrust::raw_pointer_cast(bdd_hop_dist.data()), 0})); //TODO: This needs to be changed when multiple BDDs are in one row.
        root_indices_.resize(std::distance(root_indices_.begin(), last_root));

        bot_sink_indices_ = thrust::device_vector<int>(primal_variable_index_.size());
        thrust::sequence(bot_sink_indices_.begin(), bot_sink_indices_.end());

        auto last_bot_sink = thrust::remove_if(bot_sink_indices_.begin(), bot_sink_indices_.end(),
                                            not_equal_to({thrust::raw_pointer_cast(primal_variable_index_.data()), -2}));
        bot_sink_indices_.resize(std::distance(bot_sink_indices_.begin(), last_bot_sink));

        top_sink_indices_ = thrust::device_vector<int>(primal_variable_index_.size());
        thrust::sequence(top_sink_indices_.begin(), top_sink_indices_.end());

        auto last_top_sink = thrust::remove_if(top_sink_indices_.begin(), top_sink_indices_.end(),
                                            not_equal_to({thrust::raw_pointer_cast(primal_variable_index_.data()), -1}));
        top_sink_indices_.resize(std::distance(top_sink_indices_.begin(), last_top_sink));
    }

    void bdd_cuda_base::initialize_costs()
    {
        thrust::fill(cost_from_root_.begin(), cost_from_root_.end(), HIP_INF_F);
        thrust::fill(cost_from_terminal_.begin(), cost_from_terminal_.end(), HIP_INF_F);
        thrust::fill(hi_path_cost_.begin(), hi_path_cost_.end(), HIP_INF_F);
        thrust::fill(lo_path_cost_.begin(), lo_path_cost_.end(), HIP_INF_F);
    }

    struct set_var_cost_func {
        int var_index;
        float cost;
        __host__ __device__ void operator()(const thrust::tuple<int, float&> t) const
        {
            const int cur_var_index = thrust::get<0>(t);
            if(cur_var_index != var_index)
                return;
            float& hi_cost = thrust::get<1>(t);
            hi_cost = cost;
        }
    };

    void bdd_cuda_base::set_cost(const double c, const size_t var)
    {
        assert(var < nr_vars_);
        set_var_cost_func func({(int) var, (float) c / primal_variable_counts_[var]});

        auto first = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_.begin(), hi_cost_.begin()));
        auto last = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_.end(), hi_cost_.end()));

        thrust::for_each(first, last, func);
    }

    struct set_vars_costs_func {
        int* var_counts;
        float* primal_costs;
        __host__ __device__ void operator()(const thrust::tuple<int, float&> t) const
        {
            const int cur_var_index = thrust::get<0>(t);
            float& hi_cost = thrust::get<1>(t);
            hi_cost = primal_costs[cur_var_index] / var_counts[cur_var_index];
        }
    };

    template<typename COST_ITERATOR> 
    void bdd_cuda_base::set_costs(COST_ITERATOR begin, COST_ITERATOR end)
    {
        assert(std::distance(begin, end) == nr_variables());
        thrust::device_vector<float> primal_costs(begin, end);
        
        set_vars_costs_func func({primal_variable_counts_, primal_costs});
        auto first = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_.begin(), hi_cost_.begin()));
        auto last = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_.end(), hi_cost_.end()));

        thrust::for_each(first, last, func);
    }

    __global__ void forward_step(const int cur_num_bdd_nodes, const int start_offset,
        const int* const __restrict__ lo_bdd_node_index, 
        const int* const __restrict__ hi_bdd_node_index, 
        const float* const __restrict__ hi_cost,
        float* __restrict__ cost_from_root)
    {
        const int start_index = blockIdx.x * blockDim.x + threadIdx.x;
        const int num_threads = blockDim.x * gridDim.x;
        for (int bdd_idx = start_index + start_offset; bdd_idx < cur_num_bdd_nodes + start_offset; bdd_idx += num_threads) 
        {
            const int next_lo_node = lo_bdd_node_index[bdd_idx];
            if (next_lo_node < 0) // will matter when one row contains multiple BDDs, otherwise the terminal nodes are at the end anyway.
                continue; // nothing needs to be done for terminal node.

            const int next_hi_node = hi_bdd_node_index[bdd_idx];
            assert(next_hi_node >= 0);

            const float cur_c_from_root = cost_from_root[bdd_idx];
            const float cur_hi_cost = hi_cost[bdd_idx];

            // Uncoalesced writes:
            atomicMin(&cost_from_root[next_lo_node], cur_c_from_root); // TODO: Set cost_from_root to infinity before starting next iterations.
            atomicMin(&cost_from_root[next_hi_node], cur_c_from_root + cur_hi_cost);
        }
    }

    void bdd_cuda_base::forward_run()
    {
        if (forward_state_valid_)
            return;

        // Set costs of root nodes to 0:
        thrust::scatter(thrust::make_constant_iterator<float>(0.0), 
                        thrust::make_constant_iterator<float>(0.0) + root_indices_.size(),
                        root_indices_.begin(),
                        cost_from_root_.begin());

        const int num_steps = cum_nr_bdd_nodes_per_hop_dist_.size() - 1;
        int num_nodes_processed = 0;
        for (int s = 0; s < num_steps; s++)
        {
            int threadCount = 256;
            int cur_num_bdd_nodes = cum_nr_bdd_nodes_per_hop_dist_[s] - num_nodes_processed;
            int blockCount = ceil(cur_num_bdd_nodes / (float) threadCount);
            forward_step<<<blockCount, threadCount>>>(cur_num_bdd_nodes, num_nodes_processed,
                thrust::raw_pointer_cast(lo_bdd_node_index_.data()),
                thrust::raw_pointer_cast(hi_bdd_node_index_.data()),
                thrust::raw_pointer_cast(hi_cost_.data()),
                thrust::raw_pointer_cast(cost_from_root_.data()));
            num_nodes_processed += cur_num_bdd_nodes;
        }

        forward_state_valid_ = true;
        // Set costs of bot sinks to infinity:
        // thrust::scatter(thrust::make_constant_iterator<float>(HIP_INF_F), 
        //                 thrust::make_constant_iterator<float>(HIP_INF_F) + bot_sink_indices_.size(),
        //                 bot_sink_indices_.begin(), 
        //                 cost_from_root_.begin());
    }

    __global__ void backward_step(const int cur_num_bdd_nodes, const int start_offset,
        const int* const __restrict__ lo_bdd_node_index, 
        const int* const __restrict__ hi_bdd_node_index, 
        const float* const __restrict__ hi_cost,
        const float* __restrict__ cost_from_root, 
        float* __restrict__ cost_from_terminal,
        float* __restrict__ lo_path_cost, 
        float* __restrict__ hi_path_cost)
    {
        const int start_index = blockIdx.x * blockDim.x + threadIdx.x;
        const int num_threads = blockDim.x * gridDim.x;
        for (int bdd_idx = start_index + start_offset; bdd_idx < cur_num_bdd_nodes + start_offset; bdd_idx += num_threads) 
        {
            const int lo_node = lo_bdd_node_index[bdd_idx];
            const int hi_node = hi_bdd_node_index[bdd_idx];

            const bool is_lo_bot_sink = lo_bdd_node_index[lo_node] == -2;
            const bool is_hi_bot_sink = lo_bdd_node_index[hi_node] == -2;

            if (!is_lo_bot_sink && !is_hi_bot_sink)
            {
                const float next_lo_node_cost_terminal = cost_from_terminal[lo_node];
                const float next_hi_node_cost_terminal = cost_from_terminal[hi_node];

                const float cur_hi_cost_from_terminal = next_hi_node_cost_terminal + hi_cost[bdd_idx];
                cost_from_terminal[bdd_idx] = min(cur_hi_cost_from_terminal, next_lo_node_cost_terminal);

                const float cur_cost_from_root = cost_from_root[bdd_idx];
                hi_path_cost[bdd_idx] = cur_cost_from_root + cur_hi_cost_from_terminal;
                lo_path_cost[bdd_idx] = cur_cost_from_root + next_lo_node_cost_terminal;
            }

            else if(!is_lo_bot_sink)
            {
                const float next_lo_node_cost_terminal = cost_from_terminal[lo_node];
                cost_from_terminal[bdd_idx] = next_lo_node_cost_terminal;
                lo_path_cost[bdd_idx] = cost_from_root[bdd_idx] + next_lo_node_cost_terminal;
            }
            else if(!is_hi_bot_sink)
            {
                const float cur_hi_cost_from_terminal = cost_from_terminal[hi_node] + hi_cost[bdd_idx];
                cost_from_terminal[bdd_idx] = cur_hi_cost_from_terminal;
                hi_path_cost[bdd_idx] = cost_from_root[bdd_idx] + cur_hi_cost_from_terminal;
            }
            __syncthreads();
        }
    }

    void bdd_cuda_base::backward_run()
    {
        if (backward_state_valid_)
            return;

        const int num_steps = cum_nr_bdd_nodes_per_hop_dist_.size() - 2;

        // Set costs of top sinks to 0:
        thrust::scatter(thrust::make_constant_iterator<float>(0.0), 
                        thrust::make_constant_iterator<float>(0.0) + top_sink_indices_.size(),
                        top_sink_indices_.begin(), 
                        cost_from_terminal_.begin());

        for (int s = num_steps; s >= 0; s--)
        {
            int threadCount = 256;
            int start_offset = 0;
            if(s > 0)
                start_offset = cum_nr_bdd_nodes_per_hop_dist_[s - 1];

            int cur_num_bdd_nodes = cum_nr_bdd_nodes_per_hop_dist_[s] - start_offset;
            int blockCount = ceil(cur_num_bdd_nodes / (float) threadCount);
            backward_step<<<blockCount, threadCount>>>(cur_num_bdd_nodes, start_offset,
                thrust::raw_pointer_cast(lo_bdd_node_index_.data()),
                thrust::raw_pointer_cast(hi_bdd_node_index_.data()),
                thrust::raw_pointer_cast(hi_cost_.data()),
                thrust::raw_pointer_cast(cost_from_root_.data()),
                thrust::raw_pointer_cast(cost_from_terminal_.data()),
                thrust::raw_pointer_cast(lo_path_cost_.data()),
                thrust::raw_pointer_cast(hi_path_cost_.data()));
        }

        backward_state_valid_ = true;
    }

    struct tuple_min
    {
        __host__ __device__
        thrust::tuple<float, float> operator()(const thrust::tuple<float, float>& t0, const thrust::tuple<float, float>& t1)
        {
            return thrust::make_tuple(min(thrust::get<0>(t0), thrust::get<0>(t1)), min(thrust::get<1>(t0), thrust::get<1>(t1)));
        }
    };

    // Compute min-marginals by knowing primal var index and also the bdd index of each bdd node.
    // TODO: Warp aggregation or not (?) https://on-demand.gputechconf.com/gtc/2017/presentation/s7622-Kyrylo-perelygin-robust-and-scalable-cuda.pdf
    std::tuple<thrust::device_vector<int>, thrust::device_vector<int>, thrust::device_vector<float>, thrust::device_vector<float>> 
        bdd_cuda_base::min_marginals_cuda()
    {
        forward_run();
        backward_run();

        thrust::device_vector<int> primal_variable_index_sorted = primal_variable_index_;
        thrust::device_vector<int> bdd_index_sorted = bdd_index_;
        thrust::device_vector<float> lo_path_cost_sorted = lo_path_cost_;
        thrust::device_vector<float> hi_path_cost_sorted = hi_path_cost_;

        auto first_key = thrust::make_zip_iterator(thrust::make_tuple(bdd_index_sorted.begin(), primal_variable_index_sorted.begin()));
        auto last_key = thrust::make_zip_iterator(thrust::make_tuple(bdd_index_sorted.end(), primal_variable_index_sorted.end()));

        auto first_val = thrust::make_zip_iterator(thrust::make_tuple(lo_path_cost_sorted.begin(), hi_path_cost_sorted.begin()));
        thrust::sort_by_key(first_key, last_key, first_val); //TODO: Necessary? reduce_by_key does not requires sorted only that all equal elements are consecutive.

        //TODO: Allocate less memory?
        thrust::device_vector<int> min_marginal_primal_index(nr_bdd_nodes_);
        thrust::device_vector<int> min_marginal_bdd_index(nr_bdd_nodes_);
        auto first_out_key = thrust::make_zip_iterator(thrust::make_tuple(min_marginal_bdd_index.begin(), min_marginal_primal_index.begin()));

        thrust::device_vector<float> min_marginals_lo(nr_bdd_nodes_);
        thrust::device_vector<float> min_marginals_hi(nr_bdd_nodes_); 
        auto first_out_val = thrust::make_zip_iterator(thrust::make_tuple(min_marginals_lo.begin(), min_marginals_hi.begin()));

        thrust::equal_to<thrust::tuple<int, int>> binary_pred;

        auto new_end = thrust::reduce_by_key(first_key, last_key, first_val, first_out_key, first_out_val, binary_pred, tuple_min());
        const int out_size = thrust::distance(first_out_key, new_end.first);
        // output also contains terminal BDD nodes.
        assert(num_dual_variables_ == out_size - 2 * nr_bdds_);   //TODO: Keep reusing old memory.

        min_marginals_lo.resize(out_size);
        min_marginals_hi.resize(out_size);
        min_marginal_primal_index.resize(out_size);
        min_marginal_bdd_index.resize(out_size);

        return {min_marginal_primal_index, min_marginal_bdd_index, min_marginals_lo, min_marginals_hi};
    }

    two_dim_variable_array<std::array<float,2>> bdd_cuda_base::min_marginals()
    {
        thrust::device_vector<int> mm_primal_index, mm_bdd_index;
        thrust::device_vector<float> mm_0, mm_1;

        std::tie(mm_primal_index, mm_bdd_index, mm_0, mm_1) = min_marginals_cuda();

        std::vector<int> num_vars_per_bdd(num_vars_per_bdd_.size());
        thrust::copy(num_vars_per_bdd_.begin(), num_vars_per_bdd_.end(), num_vars_per_bdd.begin());

        std::vector<int> h_mm_primal_index(mm_primal_index.size());
        thrust::copy(mm_primal_index.begin(), mm_primal_index.end(), h_mm_primal_index.begin());

        std::vector<int> h_mm_bdd_index(mm_primal_index.size());
        thrust::copy(mm_bdd_index.begin(), mm_bdd_index.end(), h_mm_bdd_index.begin());

        std::vector<float> h_mm_0(mm_primal_index.size());
        thrust::copy(mm_0.begin(), mm_0.end(), h_mm_0.begin());

        std::vector<float> h_mm_1(mm_primal_index.size());
        thrust::copy(mm_1.begin(), mm_1.end(), h_mm_1.begin());

        std::vector<size_t> h_num_vars_per_bdd(num_vars_per_bdd.size());
        thrust::copy(num_vars_per_bdd.begin(), num_vars_per_bdd.end(), h_num_vars_per_bdd.begin());

        two_dim_variable_array<std::array<float,2>> min_margs(h_num_vars_per_bdd);
        int idx_1d = 2; // ignore terminal nodes.
        for(int bdd_idx=0; bdd_idx < nr_bdds(); ++bdd_idx)
        {
            for(int var = 0; var < num_vars_per_bdd[bdd_idx]; var++, idx_1d++)
            {
                assert(h_mm_primal_index[idx_1d] >= 0); // Should ignore terminal nodes.
                std::array<float,2> mm = {h_mm_0[idx_1d], h_mm_1[idx_1d]};
                min_margs(bdd_idx, var) = mm;
            }
            idx_1d += 2; // 2 terminal nodes per bdd.
        }
        return min_margs;
    }

    struct return_top_sink_costs
    {
        __host__ __device__ double operator()(const thrust::tuple<int, float>& t) const
        {
            const int primal_index = thrust::get<0>(t);
            if (primal_index != -1)
                return 0.0;
            return thrust::get<1>(t);
        }
    };

    double bdd_cuda_base::lower_bound()
    {
        forward_run();

        // Gather all BDD nodes corresponding to top_sink (i.e. primal_variable_index == -1) and sum their costs_from_root
        auto first = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_.begin(), cost_from_root_.begin()));
        auto last = thrust::make_zip_iterator(thrust::make_tuple(primal_variable_index_.end(), cost_from_root_.end()));

        return thrust::transform_reduce(first, last, return_top_sink_costs(), 0.0, thrust::plus<double>());
    }
}
