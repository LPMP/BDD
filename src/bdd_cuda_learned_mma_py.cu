#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/eigen.h>
#include <pybind11/stl.h>
#include "run_solver_util.h"
#include "bdd_cuda_learned_mma.h"
#include "incremental_mm_agreement_rounding_cuda.h"
#include "bdd_branch_instruction.h"
#include "ILP_input.h"
#include "two_dimensional_variable_array.hxx"
#include "bdd_preprocessor.h"
#include <sstream>
#include <fstream>
#include "cuda_utils.h"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

namespace py=pybind11;

using bdd_type_default = LPMP::bdd_cuda_learned_mma<float>;
using bdd_type_double = LPMP::bdd_cuda_learned_mma<double>;

template<typename REAL>
LPMP::bdd_cuda_learned_mma<REAL> create_solver(const py::bytes& s)
{
    std::istringstream ss(s);
    cereal::BinaryInputArchive archive(ss);
    LPMP::bdd_cuda_learned_mma<REAL> solver;
    archive(solver); 
    solver.init();
    return solver;
}

struct set_primal_indices {
    const unsigned long num_vars;
    __host__ __device__ int operator()(const int i)
    {
        return min(i, (int) num_vars); // terminal nodes will have primal index = num_vars.
    }
};

template<typename T>
std::vector<size_t> argsort(const std::vector<T> &array) {
    std::vector<size_t> indices(array.size());
    std::iota(indices.begin(), indices.end(), 0);
    std::sort(indices.begin(), indices.end(),
              [&array](int left, int right) -> bool {
                  // sort indices according to corresponding array element
                  return array[left] < array[right];
              });

    return indices;
}

template<typename REAL>
std::vector<float> get_constraint_matrix_coeffs(const LPMP::ILP_input& ilp, const LPMP::bdd_cuda_learned_mma<REAL>& solver)
{
    if (ilp.nr_constraints() != solver.nr_bdds())
    {
        std::cout<<"Number of constraints: "<<ilp.nr_constraints()<<", not equal to number of BDDs: "<<solver.nr_bdds()<<"\n";
        throw std::runtime_error("error");
    }
    const std::vector<size_t> bdd_to_constraint_map = solver.bdd_to_constraint_map();
    if (bdd_to_constraint_map.size() != solver.nr_bdds())
    {
        throw std::runtime_error("bdd_to_constraint_map not calculated.");
    }

    const size_t num_elements = solver.nr_layers();
    std::vector<int> var_indices_sorted(num_elements);
    std::vector<int> con_indices_sorted(num_elements);
    std::vector<int> cumm_num_vars_per_constraint(solver.nr_bdds() + 1);
    std::vector<size_t> indices(num_elements);
    { // Create COO representation for faster indexing later.
        thrust::device_vector<int> dev_primal_index = solver.get_primal_variable_index();
        const thrust::device_vector<int> dev_bdd_index = solver.get_bdd_index();
        thrust::device_vector<unsigned long> dev_indices(num_elements);
        thrust::sequence(dev_indices.begin(), dev_indices.end());

        thrust::device_vector<size_t> dev_bdd_to_constraint_map(bdd_to_constraint_map.begin(), bdd_to_constraint_map.end());
        thrust::device_vector<int> dev_con_index(dev_bdd_index.size());

        // Map bdd_index to constraint index:
        thrust::gather(dev_bdd_index.begin(), dev_bdd_index.end(), dev_bdd_to_constraint_map.begin(), dev_con_index.begin());

        auto first_key = thrust::make_zip_iterator(thrust::make_tuple(dev_con_index.begin(), dev_primal_index.begin()));
        auto last_key = thrust::make_zip_iterator(thrust::make_tuple(dev_con_index.end(), dev_primal_index.end()));
        thrust::sort_by_key(thrust::device, first_key, last_key, dev_indices.begin());

        thrust::device_vector<int> dev_cumm_num_vars_per_constraint(num_elements);
        auto new_last = thrust::reduce_by_key(dev_con_index.begin(), dev_con_index.end(), thrust::make_constant_iterator<int>(1), 
                                thrust::make_discard_iterator(), dev_cumm_num_vars_per_constraint.begin());
        const auto nr_con = std::distance(dev_cumm_num_vars_per_constraint.begin(), new_last.second);
        if (nr_con != solver.nr_bdds())
            throw std::runtime_error("con_indices reduced size mismatch.");
        dev_cumm_num_vars_per_constraint.resize(nr_con);
        thrust::inclusive_scan(dev_cumm_num_vars_per_constraint.begin(), dev_cumm_num_vars_per_constraint.end(), 
                                dev_cumm_num_vars_per_constraint.begin());
        thrust::copy(dev_con_index.begin(), dev_con_index.end(), con_indices_sorted.begin());
        thrust::copy(dev_primal_index.begin(), dev_primal_index.end(), var_indices_sorted.begin());
        thrust::copy(dev_indices.begin(), dev_indices.end(), indices.begin());
        thrust::copy(dev_cumm_num_vars_per_constraint.begin(), dev_cumm_num_vars_per_constraint.end(), 
                    cumm_num_vars_per_constraint.begin() + 1);
        cumm_num_vars_per_constraint[0] = 0;
    }

    std::vector<float> coefficients(num_elements, 0.0);
    int find_start_index = cumm_num_vars_per_constraint[0];
    for(size_t c = 0; c < ilp.nr_constraints(); ++c)
    {
        const auto& constr = ilp.constraints()[c];
        if(!constr.is_linear())
            throw std::runtime_error("Only linear constraints supported");
        assert(constr.monomials.size() == constr.coefficients.size());
        int find_end_index = cumm_num_vars_per_constraint[c + 1];
        for(size_t monomial_idx = 0; monomial_idx < constr.monomials.size(); ++monomial_idx)
        {
            const size_t var = constr.monomials(monomial_idx, 0);
            const int coeff = constr.coefficients[monomial_idx];
            // Find where does (c, var) occurs in solver variable and constraint indices:
            const auto it = std::find(var_indices_sorted.begin() + find_start_index, 
                                    var_indices_sorted.begin() + find_end_index, var);
            if (it == var_indices_sorted.begin() + find_end_index)
            {
                std::cout<<"ILP variable not found in BDD. Var: " + std::to_string(var)<<", Con: "<<c<<"\n";
                throw std::runtime_error("error");
            }
            else
            {
                const int index_to_place = indices[std::distance(var_indices_sorted.begin(), it)];
                coefficients[index_to_place] = coeff;
            }
        }
        find_start_index = find_end_index;
    }

    return coefficients;
}

template<typename REAL>
LPMP::bdd_cuda_learned_mma<REAL>* initialize_from_ilp(const LPMP::ILP_input& ilp, const bool compute_bdd_to_constraint_map, const double objective_multiplier) 
{
    LPMP::bdd_preprocessor bdd_pre;
    const auto constraint_to_bdd_map = bdd_pre.add_ilp(ilp);
    auto* base = new LPMP::bdd_cuda_learned_mma<REAL>(bdd_pre.get_bdd_collection());
    std::vector<double> rescaled_objective = ilp.objective();
    if (objective_multiplier != 1.0)
    {
        for (int i = 0; i != rescaled_objective.size(); ++i)
            rescaled_objective[i] = objective_multiplier * rescaled_objective[i];
    }

    base->update_costs(rescaled_objective.begin(), rescaled_objective.begin(), rescaled_objective.begin(), rescaled_objective.end());
    if (compute_bdd_to_constraint_map)
        base->compute_bdd_to_constraint_map(constraint_to_bdd_map);
    return base;
}

template<typename REAL>
void lower_bound_per_bdd(LPMP::bdd_cuda_learned_mma<REAL>& solver, const long lb_out_ptr)
{
    thrust::device_ptr<REAL> lb_out_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(lb_out_ptr));
    solver.lower_bound_per_bdd(lb_out_ptr_thrust);
}

template<typename REAL>
void solution_per_bdd(LPMP::bdd_cuda_learned_mma<REAL>& solver, const long sol_out_ptr)
{
    thrust::device_ptr<REAL> sol_out_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(sol_out_ptr));
    solver.bdds_solution_cuda(sol_out_ptr_thrust);
}

template<typename REAL>
void terminal_layer_indices(LPMP::bdd_cuda_learned_mma<REAL>& solver, const long indices_out_ptr)
{
    thrust::device_ptr<int> indices_out_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<int*>(indices_out_ptr));
    solver.terminal_layer_indices(indices_out_ptr_thrust);
}

template<typename REAL>
void primal_variable_index(LPMP::bdd_cuda_learned_mma<REAL>& solver, const long primal_variable_index_out_ptr)
{
    int* ptr = reinterpret_cast<int*>(primal_variable_index_out_ptr); 
    const thrust::device_vector<int> primal_index_managed = solver.get_primal_variable_index();
    thrust::transform(primal_index_managed.begin(), primal_index_managed.end(), ptr, set_primal_indices({solver.nr_variables()}));
}

template<typename REAL>
void bdd_index(LPMP::bdd_cuda_learned_mma<REAL>& solver, const long bdd_index_out_ptr)
{
    int* ptr = reinterpret_cast<int*>(bdd_index_out_ptr); 
    const thrust::device_vector<int> bdd_index_managed = solver.get_bdd_index();
    thrust::copy(bdd_index_managed.begin(), bdd_index_managed.end(), ptr);
}

template<typename REAL>
void get_primal_objective_vector(LPMP::bdd_cuda_learned_mma<REAL>& solver, const long primal_obj_out_ptr)
{
    thrust::device_ptr<REAL> primal_obj_out_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(primal_obj_out_ptr));
    solver.compute_primal_objective_vec(primal_obj_out_ptr_thrust);
}

template<typename REAL>
void get_solver_costs(const LPMP::bdd_cuda_learned_mma<REAL>& solver, const long lo_cost_out_ptr, const long hi_cost_out_ptr, const long deferred_mm_out_ptr)
{
    thrust::device_ptr<REAL> lo_cost_out_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(lo_cost_out_ptr));
    thrust::device_ptr<REAL> hi_cost_out_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(hi_cost_out_ptr));
    thrust::device_ptr<REAL> deferred_mm_out_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(deferred_mm_out_ptr));
    solver.get_solver_costs(lo_cost_out_ptr_thrust, hi_cost_out_ptr_thrust, deferred_mm_out_ptr_thrust);
}

template<typename REAL>
void set_solver_costs(LPMP::bdd_cuda_learned_mma<REAL>& solver, const long lo_cost_ptr, const long hi_cost_ptr, const long def_mm_ptr)
{
    thrust::device_ptr<REAL> lo_cost_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(lo_cost_ptr));
    thrust::device_ptr<REAL> hi_cost_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(hi_cost_ptr));
    thrust::device_ptr<REAL> def_mm_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(def_mm_ptr));
    solver.set_solver_costs(lo_cost_ptr_thrust, hi_cost_ptr_thrust, def_mm_ptr_thrust);
}

template<typename REAL>
void non_learned_iterations(LPMP::bdd_cuda_learned_mma<REAL>& solver, const float omega, const int max_num_itr, const float improvement_slope, const float time_limit) 
{
    run_solver(solver, max_num_itr, 0.0, improvement_slope, time_limit);
}

template<typename REAL>
int iterations(LPMP::bdd_cuda_learned_mma<REAL>& solver, const long dist_weights_ptr, const int num_itr, 
                const float omega_scalar, const double improvement_slope, const long omega_vec_ptr,
                const bool omega_vec_valid, const int compute_history_for_itr, const float beta,
                const long sol_avg_ptr, const long lb_first_order_avg_ptr, const long lb_second_order_avg_ptr) 
{
    thrust::device_ptr<REAL> distw_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(dist_weights_ptr));
    thrust::device_ptr<REAL> omega_vec_thrust, sol_avg_ptr_thrust, lb_first_ptr_thrust, lb_second_ptr_thrust;
    if (compute_history_for_itr)
    {
        sol_avg_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(sol_avg_ptr)); 
        lb_first_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(lb_first_order_avg_ptr)); 
        lb_second_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(lb_second_order_avg_ptr)); 
    }
    
    if (omega_vec_valid)
    {
        omega_vec_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(omega_vec_ptr));
        return solver.iterations(distw_ptr_thrust, num_itr, 1.0, improvement_slope, 
                                sol_avg_ptr_thrust, lb_first_ptr_thrust, lb_second_ptr_thrust,
                                compute_history_for_itr, beta, omega_vec_thrust);
    }
    else
        return solver.iterations(distw_ptr_thrust, num_itr, omega_scalar, improvement_slope, 
                                sol_avg_ptr_thrust, lb_first_ptr_thrust, lb_second_ptr_thrust, 
                                compute_history_for_itr, beta);
}

template<typename REAL>
void grad_iterations(LPMP::bdd_cuda_learned_mma<REAL>& solver, 
                    const long dist_weights_ptr, const long grad_lo_cost_ptr, const long grad_hi_cost_ptr,
                    const long grad_mm_ptr, const long grad_dist_weights_out_ptr, const long grad_omega_out_ptr,
                    const float omega_scalar, const int track_grad_after_itr, const int track_grad_for_num_itr,
                    const long omega_vec_ptr, const bool omega_vec_valid, const int num_caches) 
{
    thrust::device_ptr<const REAL> dist_weights_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(dist_weights_ptr));
    thrust::device_ptr<REAL> grad_lo_cost_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_lo_cost_ptr));
    thrust::device_ptr<REAL> grad_hi_cost_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_hi_cost_ptr));
    thrust::device_ptr<REAL> grad_mm_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_mm_ptr));
    thrust::device_ptr<REAL> grad_dist_weights_out_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_dist_weights_out_ptr));
    thrust::device_ptr<REAL> grad_omega_out_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_omega_out_ptr));
    if (omega_vec_valid)
    {
        thrust::device_ptr<REAL> omega_vec_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(omega_vec_ptr));
        solver.grad_iterations(dist_weights_ptr_thrust, grad_lo_cost_ptr_thrust, grad_hi_cost_ptr_thrust,
                        grad_mm_ptr_thrust,grad_dist_weights_out_ptr_thrust, grad_omega_out_ptr_thrust,
                        1.0, track_grad_after_itr, track_grad_for_num_itr, num_caches, omega_vec_thrust);
    }
    else
        solver.grad_iterations(dist_weights_ptr_thrust, grad_lo_cost_ptr_thrust, grad_hi_cost_ptr_thrust,
                        grad_mm_ptr_thrust, grad_dist_weights_out_ptr_thrust, grad_omega_out_ptr_thrust,
                        omega_scalar, track_grad_after_itr, track_grad_for_num_itr, num_caches);
}

template<typename REAL>
void distribute_delta(LPMP::bdd_cuda_learned_mma<REAL>& solver)
{
    solver.distribute_delta();
}

template<typename REAL>
void grad_distribute_delta(LPMP::bdd_cuda_learned_mma<REAL>& solver, const long grad_lo_cost_ptr,
                        const long grad_hi_cost_ptr, const long grad_def_mm_out_ptr)
{
    thrust::device_ptr<REAL> grad_lo_cost_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_lo_cost_ptr));
    thrust::device_ptr<REAL> grad_hi_cost_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_hi_cost_ptr));
    thrust::device_ptr<REAL> grad_def_mm_out_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_def_mm_out_ptr));
    solver.grad_distribute_delta(grad_lo_cost_ptr_thrust, grad_hi_cost_ptr_thrust, grad_def_mm_out_ptr_thrust);
}

template<typename REAL>
void grad_lower_bound_per_bdd(LPMP::bdd_cuda_learned_mma<REAL>& solver, const long grad_lb_per_bdd, const long grad_lo_cost_ptr, const long grad_hi_cost_ptr)
{
    thrust::device_ptr<REAL> grad_lb_per_bdd_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_lb_per_bdd));
    thrust::device_ptr<REAL> grad_lo_cost_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_lo_cost_ptr));
    thrust::device_ptr<REAL> grad_hi_cost_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_hi_cost_ptr));
    solver.grad_lower_bound_per_bdd(grad_lb_per_bdd_thrust, grad_lo_cost_ptr_thrust, grad_hi_cost_ptr_thrust);
}

template<typename REAL>
void all_min_marginal_differences(LPMP::bdd_cuda_learned_mma<REAL>& solver,const long mm_diff_out_ptr)
{
    thrust::device_ptr<REAL> mm_diff_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(mm_diff_out_ptr));
    const auto mms = solver.min_marginals_cuda(false);
    const auto& mms_0 = std::get<1>(mms);
    const auto& mms_1 = std::get<2>(mms);
    thrust::transform(mms_1.begin(), mms_1.end(), mms_0.begin(), mm_diff_ptr_thrust, thrust::minus<double>());
}

template<typename REAL>
void grad_all_min_marginal_differences(LPMP::bdd_cuda_learned_mma<REAL>& solver, const long grad_mm_diff, 
                                    const long grad_lo_out, const long grad_hi_out)
{
    thrust::device_ptr<REAL> grad_mm_diff_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_mm_diff));
    thrust::device_ptr<REAL> grad_lo_out_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_lo_out));
    thrust::device_ptr<REAL> grad_hi_out_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_hi_out));
    solver.grad_mm_diff_all_hops(grad_mm_diff_thrust, grad_lo_out_thrust, grad_hi_out_thrust);
}

template<typename REAL>
void grad_cost_perturbation(LPMP::bdd_cuda_learned_mma<REAL>& solver,
            const long grad_lo_cost_ptr, const long grad_hi_cost_ptr,
            const long grad_lo_pert_out, const long grad_hi_pert_out)
{
    thrust::device_ptr<REAL> grad_lo_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_lo_cost_ptr));
    thrust::device_ptr<REAL> grad_hi_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_hi_cost_ptr));
    thrust::device_ptr<REAL> grad_lo_pert_out_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_lo_pert_out));
    thrust::device_ptr<REAL> grad_hi_pert_out_thrust = thrust::device_pointer_cast(reinterpret_cast<REAL*>(grad_hi_pert_out));
    solver.grad_cost_perturbation(grad_lo_thrust, grad_hi_thrust, grad_lo_pert_out_thrust, grad_hi_pert_out_thrust);
}

template<typename REAL>
std::vector<float> primal_rounding_incremental_iteration(LPMP::bdd_cuda_learned_mma<REAL>& solver, double cur_delta, const bool verbose = false)
{
    std::vector<char> sol = perturb_primal_costs(solver, cur_delta, verbose);
    std::vector<float> solution_f(sol.size());
    for (int i = 0; i < sol.size(); i++)
        solution_f[i] = (float) sol[i];
    return solution_f;
}

template<typename REAL>
std::vector<float> primal_rounding_incremental(LPMP::bdd_cuda_learned_mma<REAL>& solver, double init_delta, const double delta_growth_rate, const int num_itr_lb, const bool verbose = false, const int num_rounds = 500)
{
    std::vector<char> sol = incremental_mm_agreement_rounding_cuda(solver, init_delta, delta_growth_rate, num_itr_lb, verbose, num_rounds);
    std::vector<float> solution_f(sol.size());
    for (int i = 0; i < sol.size(); i++)
        solution_f[i] = (float) sol[i];
    return solution_f;
}

PYBIND11_MODULE(bdd_cuda_learned_mma_py, m) {
    m.doc() = "Python binding for bdd-based solver using CUDA."
            "\nbdd_cuda_learned_mma class uses FP32 for calculations."
            "\nbdd_cuda_learned_mma_double class uses FP64 for calculations.";

    py::class_<bdd_type_default>(m, "bdd_cuda_learned_mma")
        .def(py::pickle(
                    [](const bdd_type_default& solver) {
                        std::stringstream ss;
                        cereal::BinaryOutputArchive archive(ss);
                        archive(solver);
                        return py::bytes(ss.str());
                },
                    [](const py::bytes& s) {
                        return create_solver<float>(s);
                }))
        .def(py::init([](const LPMP::ILP_input& ilp, const bool compute_bdd_to_constraint_map = true, const double objective_multiplier = 1.0) 
        {
            return initialize_from_ilp<float>(ilp, compute_bdd_to_constraint_map, objective_multiplier);
        }))
        .def("__repr__", [](const bdd_type_default &solver) {
            return std::string("<bdd_cuda_learned_mma>: ") + 
                "nr_variables: "+ std::to_string(solver.nr_variables()) +
                ", nr_bdds: "+ std::to_string(solver.nr_bdds()) +
                ", nr_layers: "+ std::to_string(solver.nr_layers());
                })
        .def("export_ss", [](const bdd_type_default& solver, const std::string output_path){
            std::ofstream os(output_path, std::ios::binary);
            cereal::BinaryOutputArchive archive(os);
            archive(solver);
            std::cout<<"Exported solver data to path: "<<output_path<<"\n";
        })
        .def("nr_primal_variables", [](const bdd_type_default& solver) { return solver.nr_variables(); })
        .def("nr_layers", [](const bdd_type_default& solver) { return solver.nr_layers(); })
        .def("nr_layers", [](const bdd_type_default& solver, const int hop_index) { return solver.nr_layers(hop_index); })
        .def("nr_bdds", [](const bdd_type_default& solver) { return solver.nr_bdds(); })
        .def("constraint_matrix_coeffs", [](const bdd_type_default& solver, const LPMP::ILP_input& ilp)
        {
            return get_constraint_matrix_coeffs(ilp, solver);
        }, "Computes the coefficients for each variable appearing in constraint."
        "\nAssumes that each BDD correspond to a linear constraint present in original ILP.")
        .def("bdd_to_constraint_map", &bdd_type_default::bdd_to_constraint_map)
        .def("lower_bound", &bdd_type_default::lower_bound)
        .def("lower_bound_per_bdd", [](bdd_type_default& solver, const long lb_out_ptr)
        {
            lower_bound_per_bdd(solver, lb_out_ptr);
        }, "Computes LB for each constraint and copies in the provided pointer to FP32 memory (size = nr_bdds()).")
        .def("solution_per_bdd", [](bdd_type_default& solver, const long sol_out_ptr)
        {
            solution_per_bdd(solver, sol_out_ptr);
        }, "Computes argmin for each constraint and copies in the provided pointer to FP32 memory (size = nr_layers()).")
        .def("terminal_layer_indices", [](bdd_type_default& solver, const long indices_out_ptr)
        {
            terminal_layer_indices(solver, indices_out_ptr);
        }, "Computes indices of dual variables which are actually just terminal nodes. Input argument to point to a INT32 memory of size = nr_bdds().")
        .def("primal_variable_index", [](bdd_type_default& solver, const long primal_variable_index_out_ptr)
        {
            primal_variable_index(solver, primal_variable_index_out_ptr);
        }, "Sets primal variables indices for all dual variables in the pre-allocated memory of size = nr_layers() pointed to by the input pointer in INT32 format.\n"
        "Also contains entries for root/terminal nodes for which the values are equal to nr_variables().")
        
        .def("bdd_index", [](bdd_type_default& solver, const long bdd_index_out_ptr)
        {
            bdd_index(solver, bdd_index_out_ptr);
        }, "Sets BDD indices for all dual variables in the pre-allocated memory of size = nr_layers() pointed to by the input pointer in INT32 format.")

        .def("get_primal_objective_vector", [](bdd_type_default& solver, const long primal_obj_out_ptr)
        {
            get_primal_objective_vector(solver, primal_obj_out_ptr);
        }, "Computes primal objective vector from dual variables in the pre-allocated memory of size = nr_primal_variables() pointed to by the input pointer in FP32 format.")

        .def("get_solver_costs", [](const bdd_type_default& solver, 
                                const long lo_cost_out_ptr,
                                const long hi_cost_out_ptr,
                                const long deferred_mm_out_ptr)
        {
            get_solver_costs(solver, lo_cost_out_ptr, hi_cost_out_ptr, deferred_mm_out_ptr);
        },"Get the costs i.e., (lo_costs (size = nr_layers()), hi_costs (size = nr_layers()), deferred_mm_out_ptr_thrust (size = nr_variables()), \n"
        "and set in the memory pointed to by input pointers to preallocated memory. This method can be used to restore solver state by calling set_solver_costs().")

        .def("set_solver_costs", [](bdd_type_default& solver, 
                                const long lo_cost_ptr,
                                const long hi_cost_ptr,
                                const long def_mm_ptr)
        {
            set_solver_costs(solver, lo_cost_ptr, hi_cost_ptr, def_mm_ptr);
        },"Set the costs i.e., (lo_costs (size = nr_layers()), hi_costs (size = nr_layers()), def_mm_ptr (size = nr_layers()) to set solver state.")

        .def("non_learned_iterations", [](bdd_type_default& solver, const float omega, const int max_num_itr, const float improvement_slope, const float time_limit) 
        {
            non_learned_iterations(solver, omega, max_num_itr, improvement_slope, time_limit);
        }, "Runs parallel_mma solver for a maximum of max_num_itr iterations and stops earlier if rel. improvement is less than improvement_slope.",
        py::arg("omega") = 0.5, py::arg("max_num_itr") = 1000, py::arg("improvement_slope") = 1e-6, py::arg("time_limit") = 3600)

        .def("iterations", [](bdd_type_default& solver, 
                            const long dist_weights_ptr, 
                            const int num_itr, 
                            const float omega_scalar,
                            const double improvement_slope,
                            const long omega_vec_ptr,
                            const bool omega_vec_valid,
                            const int compute_history_for_itr,
                            const float beta,
                            const long sol_avg_ptr,
                            const long lb_first_order_avg_ptr,
                            const long lb_second_order_avg_ptr) 
        {
            return iterations(solver, dist_weights_ptr, num_itr, omega_scalar, improvement_slope, omega_vec_ptr,
                omega_vec_valid, compute_history_for_itr, beta, sol_avg_ptr, lb_first_order_avg_ptr, lb_second_order_avg_ptr);
        }, "Runs solver for num_itr many iterations using distribution weights *dist_weights_ptr and sets the min-marginals to distribute in *mm_diff_ptr.\n"
        "dist_weights_ptr, mm_diff_ptr and sol_avg_ptr should point to a memory containing nr_layers() many elements in FP32 format.\n"
        "lb_first_order_avg_ptr and lb_second_order_avg_ptr should point to a memory containing nr_bdds() many elements in FP32 format.\n"
        "If omega_vec_valid == True, then omega_vec_ptr is used (size = nr_layers()) instead of omega_scalar."
        "First iteration used the deferred min-marginals in mm_diff_ptr to distribute.")

        .def("grad_iterations", [](bdd_type_default& solver, 
                                const long dist_weights_ptr,
                                const long grad_lo_cost_ptr,
                                const long grad_hi_cost_ptr,
                                const long grad_mm_ptr,
                                const long grad_dist_weights_out_ptr,
                                const long grad_omega_out_ptr,
                                const float omega_scalar, 
                                const int track_grad_after_itr, 
                                const int track_grad_for_num_itr,
                                const long omega_vec_ptr,
                                const bool omega_vec_valid,
                                const int num_caches) 
        {
            grad_iterations(solver, dist_weights_ptr, grad_lo_cost_ptr,  grad_hi_cost_ptr,
                            grad_mm_ptr, grad_dist_weights_out_ptr,  grad_omega_out_ptr,
                            omega_scalar, track_grad_after_itr, track_grad_for_num_itr,
                            omega_vec_ptr, omega_vec_valid, num_caches);
        }, "Implements backprop through iterations().\n"
            "dist_weights: distribution weights used in the forward pass.\n"
            "grad_lo_cost: Input: incoming grad w.r.t lo_cost which were output from iterations and Outputs in-place to compute grad. lo_cost before iterations.\n"
            "grad_hi_cost: Input: incoming grad w.r.t hi_cost which were output from iterations and Outputs in-place to compute grad. hi_cost before iterations.\n"
            "grad_mm: Input: incoming grad w.r.t min-marg. diff. which were output from iterations and Outputs in-place to compute grad. w.r.t deferred min-marginals used in iterations.\n"
            "grad_dist_weights_out: Output: contains grad w.r.t distribution weights, assumes the memory is already allocated (= nr_layers()).\n"
            "grad_omega_out_ptr:  Output: contains grad w.r.t omega (size = 1)."
            "omega: floating point scalar in [0, 1] to scale current min-marginal difference before subtracting. (Same value as used in forward pass).\n"
            "track_grad_after_itr: First runs the solver for track_grad_after_itr many iterations without tracking gradients and then backpropagates through only last track_grad_for_num_itr many itrs.\n"
            "track_grad_for_num_itr: See prev. argument.\n"
            "omega_vec_ptr: vector-valued (size = nr_layers()) damping weights used in forward pass (if omega_vec_valid == True, otherwise not used).\n")

        .def("distribute_delta", [](bdd_type_default& solver) 
        {
            distribute_delta(solver);
        }, "Distributes the deferred min-marginals back to lo and hi costs such that dual constraint are satisfied with equality.\n"
            "deferred min-marginals are zero-ed out after distributing.")

        .def("grad_distribute_delta", [](bdd_type_default& solver, 
            const long grad_lo_cost_ptr,
            const long grad_hi_cost_ptr,
            const long grad_def_mm_out_ptr)
        {
            grad_distribute_delta(solver, grad_lo_cost_ptr, grad_hi_cost_ptr, grad_def_mm_out_ptr);
        }, "Backprop. through distribute_delta.")
        
        .def("grad_lower_bound_per_bdd", [](bdd_type_default& solver, const long grad_lb_per_bdd, const long grad_lo_cost_ptr, const long grad_hi_cost_ptr)
        {
            grad_lower_bound_per_bdd(solver, grad_lb_per_bdd, grad_lo_cost_ptr, grad_hi_cost_ptr);
        }, "Backprop. through lower bound per BDD.")
        
        .def("all_min_marginal_differences", [](bdd_type_default& solver, const long mm_diff_out_ptr)
        {
            all_min_marginal_differences(solver, mm_diff_out_ptr);
        }, "Computes min-marginal differences = (m^1 - m^0) for ALL dual variables and sets in memory pointed to by *mm_diff_out_ptr.")

        .def("grad_all_min_marginal_differences", [](bdd_type_default& solver, 
            const long grad_mm_diff, const long grad_lo_out, const long grad_hi_out)
        {
            grad_all_min_marginal_differences(solver, grad_mm_diff, grad_lo_out, grad_hi_out);
        }, "Computes gradient of all_min_marginal_differences().\n"
            "Receives grad. w.r.t output of all_min_marginal_differences() and computes grad_lo_cost, grad_hi_cost"
            "and sets the gradient in the memory pointed by the input pointers.")

        .def("perturb_costs", [](bdd_type_default& solver, const long lo_pert_ptr, const long hi_pert_ptr)
        {
            thrust::device_ptr<float> lo_pert_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<float*>(lo_pert_ptr));
            thrust::device_ptr<float> hi_pert_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<float*>(hi_pert_ptr));
            solver.update_costs<float>(lo_pert_ptr_thrust, solver.nr_variables(), hi_pert_ptr_thrust, solver.nr_variables());
        }, "Perturb primal costs by memory pointed by lo_pert_ptr, hi_pert_ptr. Where both inputs should point to a memory of size nr_variables().")

        .def("grad_cost_perturbation", [](bdd_type_default& solver, 
            const long grad_lo_cost_ptr, const long grad_hi_cost_ptr,
            const long grad_lo_pert_out, const long grad_hi_pert_out)
        {
            grad_cost_perturbation(solver, grad_lo_cost_ptr, grad_hi_cost_ptr, grad_lo_pert_out, grad_hi_pert_out);
        }, "During primal rounding calling update_costs(lo_pert, hi_pert) changes the dual costs, the underlying primal objective vector also changes.\n"
            "Here we compute gradients of such pertubation operation assuming that distribution of (lo_pert, hi_pert) was done with isoptropic weights.")

        .def("primal_rounding_incremental_iteration", [](bdd_type_default& solver, double cur_delta, const bool verbose)
        {
            return primal_rounding_incremental_iteration(solver, cur_delta, verbose);
        })

        .def("primal_rounding_incremental", [](bdd_type_default& solver, const int num_rounds, double init_delta, const double delta_growth_rate, const int num_itr_lb, const bool verbose)
        {
            return primal_rounding_incremental(solver, init_delta, delta_growth_rate, num_itr_lb, verbose, num_rounds);
        })
        
        .def("print_bdd_info", [](bdd_type_default& solver)
        {
            return solver.print_num_bdd_nodes_per_hop();
        });

        py::class_<bdd_type_double>(m, "bdd_cuda_learned_mma_double")
        .def(py::pickle(
                    [](const bdd_type_double& solver) {
                        std::stringstream ss;
                        cereal::BinaryOutputArchive archive(ss);
                        archive(solver);
                        return py::bytes(ss.str());
                },
                    [](const py::bytes& s) {
                        return create_solver<double>(s);
                }))
        .def(py::init([](const LPMP::ILP_input& ilp, const bool compute_bdd_to_constraint_map = true, const double objective_multiplier = 1.0) 
        {
            return initialize_from_ilp<double>(ilp, compute_bdd_to_constraint_map, objective_multiplier);
        }))
        .def("__repr__", [](const bdd_type_double &solver) {
            return std::string("<bdd_cuda_learned_mma>: ") + 
                "nr_variables: "+ std::to_string(solver.nr_variables()) +
                ", nr_bdds: "+ std::to_string(solver.nr_bdds()) +
                ", nr_layers: "+ std::to_string(solver.nr_layers());
                })
        .def("export_ss", [](const bdd_type_double& solver, const std::string output_path){
            std::ofstream os(output_path, std::ios::binary);
            cereal::BinaryOutputArchive archive(os);
            archive(solver);
            std::cout<<"Exported solver data to path: "<<output_path<<"\n";
        })
        .def("nr_primal_variables", [](const bdd_type_double& solver) { return solver.nr_variables(); })
        .def("nr_layers", [](const bdd_type_double& solver) { return solver.nr_layers(); })
        .def("nr_layers", [](const bdd_type_double& solver, const int hop_index) { return solver.nr_layers(hop_index); })
        .def("nr_bdds", [](const bdd_type_double& solver) { return solver.nr_bdds(); })
        .def("constraint_matrix_coeffs", [](const bdd_type_double& solver, const LPMP::ILP_input& ilp)
        {
            return get_constraint_matrix_coeffs(ilp, solver);
        }, "Computes the coefficients for each variable appearing in constraint."
        "\nAssumes that each BDD correspond to a linear constraint present in original ILP.")
        .def("bdd_to_constraint_map", &bdd_type_double::bdd_to_constraint_map)
        .def("lower_bound", &bdd_type_double::lower_bound)
        .def("lower_bound_per_bdd", [](bdd_type_double& solver, const long lb_out_ptr)
        {
            lower_bound_per_bdd(solver, lb_out_ptr);
        }, "Computes LB for each constraint and copies in the provided pointer to FP32 memory (size = nr_bdds()).")
        .def("solution_per_bdd", [](bdd_type_double& solver, const long sol_out_ptr)
        {
            solution_per_bdd(solver, sol_out_ptr);
        }, "Computes argmin for each constraint and copies in the provided pointer to FP32 memory (size = nr_layers()).")
        .def("terminal_layer_indices", [](bdd_type_double& solver, const long indices_out_ptr)
        {
            terminal_layer_indices(solver, indices_out_ptr);
        }, "Computes indices of dual variables which are actually just terminal nodes. Input argument to point to a INT32 memory of size = nr_bdds().")
        .def("primal_variable_index", [](bdd_type_double& solver, const long primal_variable_index_out_ptr)
        {
            primal_variable_index(solver, primal_variable_index_out_ptr);
        }, "Sets primal variables indices for all dual variables in the pre-allocated memory of size = nr_layers() pointed to by the input pointer in INT32 format.\n"
        "Also contains entries for root/terminal nodes for which the values are equal to nr_variables().")
        
        .def("bdd_index", [](bdd_type_double& solver, const long bdd_index_out_ptr)
        {
            bdd_index(solver, bdd_index_out_ptr);
        }, "Sets BDD indices for all dual variables in the pre-allocated memory of size = nr_layers() pointed to by the input pointer in INT32 format.")

        .def("get_primal_objective_vector", [](bdd_type_double& solver, const long primal_obj_out_ptr)
        {
            get_primal_objective_vector(solver, primal_obj_out_ptr);
        }, "Computes primal objective vector from dual variables in the pre-allocated memory of size = nr_primal_variables() pointed to by the input pointer in FP32 format.")

        .def("get_solver_costs", [](const bdd_type_double& solver, 
                                const long lo_cost_out_ptr,
                                const long hi_cost_out_ptr,
                                const long deferred_mm_out_ptr)
        {
            get_solver_costs(solver, lo_cost_out_ptr, hi_cost_out_ptr, deferred_mm_out_ptr);
        },"Get the costs i.e., (lo_costs (size = nr_layers()), hi_costs (size = nr_layers()), deferred_mm_out_ptr_thrust (size = nr_variables()), \n"
        "and set in the memory pointed to by input pointers to preallocated memory. This method can be used to restore solver state by calling set_solver_costs().")

        .def("set_solver_costs", [](bdd_type_double& solver, 
                                const long lo_cost_ptr,
                                const long hi_cost_ptr,
                                const long def_mm_ptr)
        {
            set_solver_costs(solver, lo_cost_ptr, hi_cost_ptr, def_mm_ptr);
        },"Set the costs i.e., (lo_costs (size = nr_layers()), hi_costs (size = nr_layers()), def_mm_ptr (size = nr_layers()) to set solver state.")

        .def("non_learned_iterations", [](bdd_type_double& solver, const float omega, const int max_num_itr, const float improvement_slope, const float time_limit) 
        {
            non_learned_iterations(solver, omega, max_num_itr, improvement_slope, time_limit);
        }, "Runs parallel_mma solver for a maximum of max_num_itr iterations and stops earlier if rel. improvement is less than improvement_slope.",
        py::arg("omega") = 0.5, py::arg("max_num_itr") = 1000, py::arg("improvement_slope") = 1e-6, py::arg("time_limit") = 3600)

        .def("iterations", [](bdd_type_double& solver, 
                            const long dist_weights_ptr, 
                            const int num_itr, 
                            const float omega_scalar,
                            const double improvement_slope,
                            const long omega_vec_ptr,
                            const bool omega_vec_valid,
                            const int compute_history_for_itr,
                            const float beta,
                            const long sol_avg_ptr,
                            const long lb_first_order_avg_ptr,
                            const long lb_second_order_avg_ptr) 
        {
            return iterations(solver, dist_weights_ptr, num_itr, omega_scalar, improvement_slope, omega_vec_ptr,
                omega_vec_valid, compute_history_for_itr, beta, sol_avg_ptr, lb_first_order_avg_ptr, lb_second_order_avg_ptr);
        }, "Runs solver for num_itr many iterations using distribution weights *dist_weights_ptr and sets the min-marginals to distribute in *mm_diff_ptr.\n"
        "dist_weights_ptr, mm_diff_ptr and sol_avg_ptr should point to a memory containing nr_layers() many elements in FP32 format.\n"
        "lb_first_order_avg_ptr and lb_second_order_avg_ptr should point to a memory containing nr_bdds() many elements in FP32 format.\n"
        "If omega_vec_valid == True, then omega_vec_ptr is used (size = nr_layers()) instead of omega_scalar."
        "First iteration used the deferred min-marginals in mm_diff_ptr to distribute.")

        .def("grad_iterations", [](bdd_type_double& solver, 
                                const long dist_weights_ptr,
                                const long grad_lo_cost_ptr,
                                const long grad_hi_cost_ptr,
                                const long grad_mm_ptr,
                                const long grad_dist_weights_out_ptr,
                                const long grad_omega_out_ptr,
                                const float omega_scalar, 
                                const int track_grad_after_itr, 
                                const int track_grad_for_num_itr,
                                const long omega_vec_ptr,
                                const bool omega_vec_valid,
                                const int num_caches) 
        {
            grad_iterations(solver, dist_weights_ptr, grad_lo_cost_ptr,  grad_hi_cost_ptr,
                            grad_mm_ptr, grad_dist_weights_out_ptr,  grad_omega_out_ptr,
                            omega_scalar, track_grad_after_itr, track_grad_for_num_itr,
                            omega_vec_ptr, omega_vec_valid, num_caches);
        }, "Implements backprop through iterations().\n"
            "dist_weights: distribution weights used in the forward pass.\n"
            "grad_lo_cost: Input: incoming grad w.r.t lo_cost which were output from iterations and Outputs in-place to compute grad. lo_cost before iterations.\n"
            "grad_hi_cost: Input: incoming grad w.r.t hi_cost which were output from iterations and Outputs in-place to compute grad. hi_cost before iterations.\n"
            "grad_mm: Input: incoming grad w.r.t min-marg. diff. which were output from iterations and Outputs in-place to compute grad. w.r.t deferred min-marginals used in iterations.\n"
            "grad_dist_weights_out: Output: contains grad w.r.t distribution weights, assumes the memory is already allocated (= nr_layers()).\n"
            "grad_omega_out_ptr:  Output: contains grad w.r.t omega (size = 1)."
            "omega: floating point scalar in [0, 1] to scale current min-marginal difference before subtracting. (Same value as used in forward pass).\n"
            "track_grad_after_itr: First runs the solver for track_grad_after_itr many iterations without tracking gradients and then backpropagates through only last track_grad_for_num_itr many itrs.\n"
            "track_grad_for_num_itr: See prev. argument.\n"
            "omega_vec_ptr: vector-valued (size = nr_layers()) damping weights used in forward pass (if omega_vec_valid == True, otherwise not used).\n")

        .def("distribute_delta", [](bdd_type_double& solver) 
        {
            distribute_delta(solver);
        }, "Distributes the deferred min-marginals back to lo and hi costs such that dual constraint are satisfied with equality.\n"
            "deferred min-marginals are zero-ed out after distributing.")

        .def("grad_distribute_delta", [](bdd_type_double& solver, 
            const long grad_lo_cost_ptr,
            const long grad_hi_cost_ptr,
            const long grad_def_mm_out_ptr)
        {
            grad_distribute_delta(solver, grad_lo_cost_ptr, grad_hi_cost_ptr, grad_def_mm_out_ptr);
        }, "Backprop. through distribute_delta.")
        
        .def("grad_lower_bound_per_bdd", [](bdd_type_double& solver, const long grad_lb_per_bdd, const long grad_lo_cost_ptr, const long grad_hi_cost_ptr)
        {
            grad_lower_bound_per_bdd(solver, grad_lb_per_bdd, grad_lo_cost_ptr, grad_hi_cost_ptr);
        }, "Backprop. through lower bound per BDD.")
        
        .def("all_min_marginal_differences", [](bdd_type_double& solver, const long mm_diff_out_ptr)
        {
            all_min_marginal_differences(solver, mm_diff_out_ptr);
        }, "Computes min-marginal differences = (m^1 - m^0) for ALL dual variables and sets in memory pointed to by *mm_diff_out_ptr.")

        .def("grad_all_min_marginal_differences", [](bdd_type_double& solver, 
            const long grad_mm_diff, const long grad_lo_out, const long grad_hi_out)
        {
            grad_all_min_marginal_differences(solver, grad_mm_diff, grad_lo_out, grad_hi_out);
        }, "Computes gradient of all_min_marginal_differences().\n"
            "Receives grad. w.r.t output of all_min_marginal_differences() and computes grad_lo_cost, grad_hi_cost"
            "and sets the gradient in the memory pointed by the input pointers.")

        .def("perturb_costs", [](bdd_type_double& solver, const long lo_pert_ptr, const long hi_pert_ptr)
        {
            thrust::device_ptr<double> lo_pert_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<double*>(lo_pert_ptr));
            thrust::device_ptr<double> hi_pert_ptr_thrust = thrust::device_pointer_cast(reinterpret_cast<double*>(hi_pert_ptr));
            solver.update_costs<double>(lo_pert_ptr_thrust, solver.nr_variables(), hi_pert_ptr_thrust, solver.nr_variables());
        }, "Perturb primal costs by memory pointed by lo_pert_ptr, hi_pert_ptr. Where both inputs should point to a memory of size nr_variables().")

        .def("grad_cost_perturbation", [](bdd_type_double& solver, 
            const long grad_lo_cost_ptr, const long grad_hi_cost_ptr,
            const long grad_lo_pert_out, const long grad_hi_pert_out)
        {
            grad_cost_perturbation(solver, grad_lo_cost_ptr, grad_hi_cost_ptr, grad_lo_pert_out, grad_hi_pert_out);
        }, "During primal rounding calling update_costs(lo_pert, hi_pert) changes the dual costs, the underlying primal objective vector also changes.\n"
            "Here we compute gradients of such pertubation operation assuming that distribution of (lo_pert, hi_pert) was done with isoptropic weights.")

        .def("primal_rounding_incremental_iteration", [](bdd_type_double& solver, double cur_delta, const bool verbose)
        {
            return primal_rounding_incremental_iteration(solver, cur_delta, verbose);
        })

        .def("primal_rounding_incremental", [](bdd_type_double& solver, const int num_rounds, double init_delta, const double delta_growth_rate, const int num_itr_lb, const bool verbose)
        {
            return primal_rounding_incremental(solver, init_delta, delta_growth_rate, num_itr_lb, verbose, num_rounds);
        })
        
        .def("print_bdd_info", [](bdd_type_double& solver)
        {
            return solver.print_num_bdd_nodes_per_hop();
        });
}

