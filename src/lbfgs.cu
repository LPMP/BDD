#include "hip/hip_runtime.h"
#include "time_measure_util.h"
#include "cuda_utils.h"
#include "lbfgs.h"
#include <thrust/for_each.h>
#include <thrust/inner_product.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

namespace LPMP {

    template<typename VECTOR, typename REAL>
    lbfgs<VECTOR, REAL>::lbfgs(const size_t _num_variables, const int _history_size, 
        const double _init_lb_increase, const double _init_step_size, const double _req_rel_lb_increase, 
        const double _step_size_decrease_factor, const double _step_size_increase_factor) : 
        num_variables(_num_variables), m(_history_size), 
        init_lb_increase(_init_lb_increase), step_size(_init_step_size), required_relative_lb_increase(_req_rel_lb_increase),
        step_size_decrease_factor(_step_size_decrease_factor), step_size_increase_factor(_step_size_increase_factor)
    {
        prev_x = VECTOR(num_variables);
        prev_grad_f = VECTOR(num_variables);
        rho_inv_history = std::vector<REAL>(m);
        s_history = std::vector<VECTOR>(m);
        y_history = std::vector<VECTOR>(m);
        std::cout<<"[lbfgs] Initialized LBFGS with history size: "<<m<<"\n";
    }

    template<typename VECTOR, typename REAL>
    template<typename SOLVER>
    void lbfgs<VECTOR, REAL>::store_iterate(const SOLVER& s)
    {
        // TODO: Provide following two functions in all solvers.
        // should return thrust::device_vector fror GPU and thrust::host_vector/std::vector for CPU solvers.
        VECTOR cur_x = s.net_solver_costs();
        VECTOR cur_grad_f = s.bdds_solution_vec();
        
        assert(cur_x.size() == prev_x.size());
        assert(cur_grad_f.size() == prev_x.size());
        if (!prev_states_stored)
        {
            prev_x = cur_x;
            prev_grad_f = cur_grad_f;
            prev_states_stored = true;
        }
        else
        {
            VECTOR cur_s(cur_x.size()); // compute x_k - x_{k-1}
            thrust::transform(cur_x.begin(), cur_x.end(), prev_x.begin(), cur_s.begin(), thrust::minus<REAL>());

            // compute grad_f_k - grad_f_{k-1}, but since we have maximization problem and lbfgs updates are derived for minimization so multiply gradients by -1.
            VECTOR cur_y(cur_grad_f.size());
            thrust::transform(prev_grad_f.begin(), prev_grad_f.end(), cur_grad_f.begin(), cur_y.begin(), thrust::minus<REAL>());

            REAL rho_inv = thrust::inner_product(cur_s.begin(), cur_s.end(), cur_y.begin(), (REAL) 0.0);
            if (!initial_rho_inv_valid)
            {
                initial_rho_inv = rho_inv;
                initial_rho_inv_valid = true;
            }
            if (rho_inv / initial_rho_inv > 1e-8) // otherwise, skip the iterate as curvature condition is not strongly satisfied.
            {
                rho_inv_history[next_insertion_index] = rho_inv;
                s_history[next_insertion_index] = cur_s;
                y_history[next_insertion_index] = cur_y;
                next_insertion_index = (next_insertion_index + 1) % m;
                num_stored = min(num_stored + 1, m);
            } // when skipping estimate of hessian will become out-of-date. However removing these updates as below gives worse results than not removing.
            else
            {
                num_stored = max(num_stored - 1, 0);
            }
            prev_x = cur_x;
            prev_grad_f = cur_grad_f;
        }
    }

    template<typename VECTOR, typename REAL>
    template<typename SOLVER>
    void lbfgs<VECTOR, REAL>::iteration(SOLVER& s)
    {
        // 1. Update LBFGS states:
        this->store_iterate(s);

        // 2. Check if enough history accumulated
       if (!this->update_possible())
            return;

        // 3. Compute LBFGS update direction. This can be infeasible.
        VECTOR grad_lbfgs = this->compute_update_direction();

        // 4. Make the update direction dual feasible by making it sum to zero for all primal variables.
        s.make_dual_feasible(grad_lbfgs.begin(), grad_lbfgs.end()); //TODO: Implement for all solvers
        
        // 5. Apply the update by choosing appropriate step size:
        this->apply_update(s, grad_lbfgs);
    }

    template<typename VECTOR, typename REAL>
    template<typename SOLVER>
    void lbfgs<VECTOR, REAL>::search_step_size_and_apply(SOLVER& s, const VECTOR& update)
    {    
        const REAL lb_pre = s.lower_bound();
        auto calculate_rel_change = [&]() {
            return (s.lower_bound() - lb_pre) / (1e-9 + this->init_lb_increase);
        };
        double prev_step_size = 0.0;
        auto apply_update = [&](const REAL new_step_size) 
        {
            double net_step_size = new_step_size - prev_step_size;
            if (net_step_size != 0.0)
                s.update_dual_costs_with_step_size(update.begin(), update.end(), net_step_size); // TODO: implement for each solver.
            prev_step_size = net_step_size;
        };

        size_t num_updates = 0;
        REAL curr_rel_change = 0.0;
        REAL best_step_size = 0.0;
        REAL best_rel_improvement = 0.0;
        do
        {
            apply_update(this->step_size);
            curr_rel_change = calculate_rel_change();
            if (best_rel_improvement < curr_rel_change)
            {
                best_rel_improvement = curr_rel_change;
                best_step_size = this->step_size;
            }
            if (curr_rel_change <= 0.0)
                this->step_size *= this->step_size_decrease_factor;
            else if (curr_rel_change < required_relative_lb_increase)
                this->step_size *= this->step_size_increase_factor;

            std::cout<<"[lbfgs] relative_change: "<<curr_rel_change<<", step size: "<<this->step_size<<"\n";
            if (num_updates > 5)
            {
                if (best_rel_improvement > required_relative_lb_increase / 10.0) //TODO: Have a separate parameter?
                    apply_update(best_step_size);
                else
                {
                    apply_update(0.0);
                    this->num_unsuccessful_lbfgs_updates_ += 1;
                }
                return;
            }
            num_updates++;
        } while(curr_rel_change < required_relative_lb_increase);
        if (num_updates == 1 && this->num_unsuccessful_lbfgs_updates_ == 0)
            this->step_size *= this->step_size_increase_factor;
        this->num_unsuccessful_lbfgs_updates_ = 0;
    }


    template<typename REAL>
    struct update_q
    {
        const REAL alpha;
        const REAL* y;
        REAL* q;
        __host__ __device__ void operator()(const int idx)
        {
            q[idx] -= alpha * y[idx];
        }
    };

    template<typename REAL>
    struct update_r
    {
        const REAL alpha;
        const REAL beta;
        const REAL* s;
        REAL* r;
        __host__ __device__ void operator()(const int idx)
        {
            r[idx] += s[idx] * (alpha - beta);
        }
    };

    template<typename VECTOR, typename REAL>
    VECTOR lbfgs<VECTOR, REAL>::compute_update_direction()
    {
        assert(this->update_possible());
        VECTOR direction(num_variables);

        const int n = s_history[0].size();

        std::vector<REAL> alpha_history;
        for (int count = 0; count < num_stored; count++)
        {
            int i = next_insertion_index - count - 1;
            if (i < 0)
                i = m + i;

            assert(i >= 0 && i < m);
            assert(s_history[i].size() == n);
            const REAL alpha = thrust::inner_product(s_history[i].begin(), s_history[i].end(), direction.begin(), (REAL) 0.0) / (rho_inv_history[i]);
            
            alpha_history.push_back(alpha);
            update_q<REAL> update_q_func({alpha, thrust::raw_pointer_cast(y_history[i].data()), thrust::raw_pointer_cast(direction.data())});

            thrust::for_each(thrust::make_counting_iterator<int>(0), thrust::make_counting_iterator<int>(0) + n, update_q_func);
        }

        REAL last_y_norm = thrust::inner_product(y_history.back().begin(), y_history.back().end(), y_history.back().begin(), (REAL) 0.0);
        REAL initial_H_diag_multiplier = rho_inv_history.back() / (1e-8 + last_y_norm);
        // Skip line 5 in Alg.1 and fuse with line 7 for first loop itr.
        for (int count = 0; count < num_stored; count++)
        {
            int i = next_insertion_index - num_stored + count;
            if (i < 0)
                i = m + i;

            assert(i >= 0 && i < m);
            assert(y_history[i].size() == n);

            REAL current_rho = 1 / (rho_inv_history[i]);
            if (count == 0)
                current_rho *= initial_H_diag_multiplier;
            const REAL beta = current_rho * thrust::inner_product(y_history[i].begin(), y_history[i].end(), direction.begin(), (REAL) 0.0);

            update_r<REAL> update_r_func({alpha_history[num_stored - count - 1], beta, thrust::raw_pointer_cast(s_history[i].data()), thrust::raw_pointer_cast(direction.data())});

            thrust::for_each(thrust::make_counting_iterator<int>(0), thrust::make_counting_iterator<int>(0) + n, update_r_func);
        }
        return direction;
    }

    template<typename VECTOR, typename REAL>
    void lbfgs<VECTOR, REAL>::flush_states()
    {
        num_stored = 0;
        next_insertion_index = 0;
        prev_states_stored = false;
        initial_rho_inv = 0.0;
        initial_rho_inv_valid = false;
    }

    template<typename VECTOR, typename REAL>
    void lbfgs<VECTOR, REAL>::next_itr_without_storage()
    {
        num_stored = max(num_stored - 1, 0);
    }

    template<typename VECTOR, typename REAL>
    bool lbfgs<VECTOR, REAL>::update_possible()
    {
        if (num_stored < m)
            return false;
        return true;
    }

    template class lbfgs<thrust::host_vector<float>, float>;
    template class lbfgs<thrust::device_vector<float>, float>;
    template class lbfgs<thrust::host_vector<double>, double>;
    template class lbfgs<thrust::device_vector<double>, double>;
}