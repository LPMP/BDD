#include "hip/hip_runtime.h"
#include "bdd_cuda_learned_mma.h"
#include "ILP_parser.h"
#include "bdd_collection/bdd_collection.h"
#include "bdd_preprocessor.h"
#include "test.h"
#include "cuda_utils.h"

using namespace LPMP;
using namespace BDD;

const char * two_simplex_non_unique_sols = 
R"(Minimize
1 x_1 + 1 x_2 + 1 x_3
+2 x_4 + 1 x_5 + 1 x_6
Subject To
x_1 + x_2 + x_3 + x_4 = 1
x_4 + x_5 + x_6 = 2
End)";

const char * matching_3x3 = 
R"(Minimize
-2 x_11 - 1 x_12 - 1 x_13
-1 x_21 - 2 x_22 - 1 x_23
-1 x_31 - 1 x_32 - 2 x_33
Subject To
x_11 + x_12 + x_13 = 1
x_21 + x_22 + x_23 = 1
x_31 + x_32 + x_33 = 1
x_11 + x_21 + x_31 = 1
x_12 + x_22 + x_32 = 1
x_13 + x_23 + x_33 = 1
End)";

const char * short_chain_shuffled = 
R"(Minimize
+ 1 mu_2_1 + 1 mu_10 + 0 mu_1_1 + 0 mu_11
-1 mu_1_0 + 1 mu_00 + 2 mu_01 + 2 mu_2_0
Subject To
mu_1_0 + mu_1_1 = 1
mu_2_0 + mu_2_1 = 1
mu_00 + mu_10 + mu_01 + mu_11 = 1
mu_1_0 - mu_00 - mu_01 = 0
mu_1_1 - mu_10 - mu_11 = 0
mu_2_0 - mu_00 - mu_10 = 0
mu_2_1 - mu_01 - mu_11 = 0
End)";

const char * long_chain = 
R"(Minimize
2 mu_0_0 - 1 mu_0_1 + 3 mu_1_0 - 1 mu_1_1
+ 3 mu_2_0 + 2 mu_2_1 - 1 mu_3_0 - 2 mu_3_1
- 2 mu_4_0 - 1 mu_4_1 + 1 mu_5_0 - 1 mu_5_1
+ 1 mu_6_0 + 1 mu_6_1 - 3 mu_7_0 + 2 mu_7_1
+ 0 mu_8_0 + 2 mu_8_1
+ 1 mu_01_00 - 2 mu_01_01 + 2 mu_01_10 - 1 mu_01_11
+ 0 mu_12_00 - 1 mu_12_01 + 1 mu_12_10 + 0 mu_12_11
- 1 mu_23_00 + 2 mu_23_01 + 1 mu_23_10 - 2 mu_23_11
+ 2 mu_34_00 + 0 mu_34_01 + 2 mu_34_10 + 2 mu_34_11
+ 1 mu_45_00 - 2 mu_45_01 - 3 mu_45_10 - 1 mu_45_11
- 2 mu_56_00 + 0 mu_56_01 + 1 mu_56_10 + 3 mu_56_11
- 1 mu_67_00 - 2 mu_67_01 - 1 mu_67_10 - 1 mu_67_11
+ 2 mu_78_00 + 0 mu_78_01 + 2 mu_78_10 + 3 mu_78_11
Subject To
mu_0_0 + mu_0_1 = 1
mu_1_0 + mu_1_1 = 1
mu_2_0 + mu_2_1 = 1
mu_3_0 + mu_3_1 = 1
mu_4_0 + mu_4_1 = 1
mu_5_0 + mu_5_1 = 1
mu_6_0 + mu_6_1 = 1
mu_7_0 + mu_7_1 = 1
mu_8_0 + mu_8_1 = 1
mu_01_00 + mu_01_10 + mu_01_01 + mu_01_11 = 1
mu_12_00 + mu_12_10 + mu_12_01 + mu_12_11 = 1
mu_23_00 + mu_23_10 + mu_23_01 + mu_23_11 = 1
mu_34_00 + mu_34_10 + mu_34_01 + mu_34_11 = 1
mu_45_00 + mu_45_10 + mu_45_01 + mu_45_11 = 1
mu_56_00 + mu_56_10 + mu_56_01 + mu_56_11 = 1
mu_67_00 + mu_67_10 + mu_67_01 + mu_67_11 = 1
mu_78_00 + mu_78_10 + mu_78_01 + mu_78_11 = 1
mu_0_0 - mu_01_00 - mu_01_01 = 0
mu_0_1 - mu_01_10 - mu_01_11 = 0
mu_1_0 - mu_01_00 - mu_01_10 = 0
mu_1_1 - mu_01_01 - mu_01_11 = 0
mu_1_0 - mu_12_00 - mu_12_01 = 0
mu_1_1 - mu_12_10 - mu_12_11 = 0
mu_2_0 - mu_12_00 - mu_12_10 = 0
mu_2_1 - mu_12_01 - mu_12_11 = 0
mu_2_0 - mu_23_00 - mu_23_01 = 0
mu_2_1 - mu_23_10 - mu_23_11 = 0
mu_3_0 - mu_23_00 - mu_23_10 = 0
mu_3_1 - mu_23_01 - mu_23_11 = 0
mu_3_0 - mu_34_00 - mu_34_01 = 0
mu_3_1 - mu_34_10 - mu_34_11 = 0
mu_4_0 - mu_34_00 - mu_34_10 = 0
mu_4_1 - mu_34_01 - mu_34_11 = 0
mu_4_0 - mu_45_00 - mu_45_01 = 0
mu_4_1 - mu_45_10 - mu_45_11 = 0
mu_5_0 - mu_45_00 - mu_45_10 = 0
mu_5_1 - mu_45_01 - mu_45_11 = 0
mu_5_0 - mu_56_00 - mu_56_01 = 0
mu_5_1 - mu_56_10 - mu_56_11 = 0
mu_6_0 - mu_56_00 - mu_56_10 = 0
mu_6_1 - mu_56_01 - mu_56_11 = 0
mu_6_0 - mu_67_00 - mu_67_01 = 0
mu_6_1 - mu_67_10 - mu_67_11 = 0
mu_7_0 - mu_67_00 - mu_67_10 = 0
mu_7_1 - mu_67_01 - mu_67_11 = 0
mu_7_0 - mu_78_00 - mu_78_01 = 0
mu_7_1 - mu_78_10 - mu_78_11 = 0
mu_8_0 - mu_78_00 - mu_78_10 = 0
mu_8_1 - mu_78_01 - mu_78_11 = 0
End)";

const char * grid_graph_3x3 = 
R"(Minimize
2 mu_0_0 - 1 mu_0_1 + 3 mu_1_0 - 1 mu_1_1
+ 3 mu_2_0 + 2 mu_2_1 - 1 mu_3_0 - 2 mu_3_1
- 2 mu_4_0 - 1 mu_4_1 + 3 mu_5_0 - 1 mu_5_1
+ 1 mu_6_0 + 1 mu_6_1 - 3 mu_7_0 + 2 mu_7_1
+ 0 mu_8_0 + 2 mu_8_1
+ 1 mu_01_00 - 2 mu_01_01 + 2 mu_01_10 - 1 mu_01_11
+ 0 mu_12_00 + 1 mu_12_01 + 1 mu_12_10 + 0 mu_12_11
- 1 mu_03_00 + 2 mu_03_01 + 0 mu_03_10 - 2 mu_03_11
+ 2 mu_14_00 + 0 mu_14_01 + 2 mu_14_10 + 2 mu_14_11
+ 1 mu_25_00 - 2 mu_25_01 - 3 mu_25_10 - 1 mu_25_11
+ 0 mu_34_00 + 1 mu_34_01 + 1 mu_34_10 + 1 mu_34_11
- 1 mu_45_00 - 2 mu_45_01 + 4 mu_45_10 - 2 mu_45_11
- 2 mu_36_00 + 0 mu_36_01 + 1 mu_36_10 + 3 mu_36_11
+ 3 mu_47_00 - 2 mu_47_01 - 2 mu_47_10 - 1 mu_47_11
+ 0 mu_58_00 + 1 mu_58_01 + 1 mu_58_10 + 1 mu_58_11
- 1 mu_67_00 + 2 mu_67_01 - 1 mu_67_10 - 1 mu_67_11
+ 2 mu_78_00 + 0 mu_78_01 + 2 mu_78_10 + 2 mu_78_11
Subject To
mu_0_0 + mu_0_1 = 1
mu_1_0 + mu_1_1 = 1
mu_2_0 + mu_2_1 = 1
mu_3_0 + mu_3_1 = 1
mu_4_0 + mu_4_1 = 1
mu_5_0 + mu_5_1 = 1
mu_6_0 + mu_6_1 = 1
mu_7_0 + mu_7_1 = 1
mu_8_0 + mu_8_1 = 1
mu_01_00 + mu_01_10 + mu_01_01 + mu_01_11 = 1
mu_12_00 + mu_12_10 + mu_12_01 + mu_12_11 = 1
mu_03_00 + mu_03_10 + mu_03_01 + mu_03_11 = 1
mu_14_00 + mu_14_10 + mu_14_01 + mu_14_11 = 1
mu_25_00 + mu_25_10 + mu_25_01 + mu_25_11 = 1
mu_34_00 + mu_34_10 + mu_34_01 + mu_34_11 = 1
mu_45_00 + mu_45_10 + mu_45_01 + mu_45_11 = 1
mu_36_00 + mu_36_10 + mu_36_01 + mu_36_11 = 1
mu_47_00 + mu_47_10 + mu_47_01 + mu_47_11 = 1
mu_58_00 + mu_58_10 + mu_58_01 + mu_58_11 = 1
mu_67_00 + mu_67_10 + mu_67_01 + mu_67_11 = 1
mu_78_00 + mu_78_10 + mu_78_01 + mu_78_11 = 1
mu_0_0 - mu_01_00 - mu_01_01 = 0
mu_0_1 - mu_01_10 - mu_01_11 = 0
mu_0_0 - mu_03_00 - mu_03_01 = 0
mu_0_1 - mu_03_10 - mu_03_11 = 0
mu_1_0 - mu_01_00 - mu_01_10 = 0
mu_1_1 - mu_01_01 - mu_01_11 = 0
mu_1_0 - mu_12_00 - mu_12_01 = 0
mu_1_1 - mu_12_10 - mu_12_11 = 0
mu_1_0 - mu_14_00 - mu_14_01 = 0
mu_1_1 - mu_14_10 - mu_14_11 = 0
mu_2_0 - mu_12_00 - mu_12_10 = 0
mu_2_1 - mu_12_01 - mu_12_11 = 0
mu_2_0 - mu_25_00 - mu_25_01 = 0
mu_2_1 - mu_25_10 - mu_25_11 = 0
mu_3_0 - mu_03_00 - mu_03_10 = 0
mu_3_1 - mu_03_01 - mu_03_11 = 0
mu_3_0 - mu_34_00 - mu_34_01 = 0
mu_3_1 - mu_34_10 - mu_34_11 = 0
mu_3_0 - mu_36_00 - mu_36_01 = 0
mu_3_1 - mu_36_10 - mu_36_11 = 0
mu_4_0 - mu_14_00 - mu_14_10 = 0
mu_4_1 - mu_14_01 - mu_14_11 = 0
mu_4_0 - mu_34_00 - mu_34_10 = 0
mu_4_1 - mu_34_01 - mu_34_11 = 0
mu_4_0 - mu_45_00 - mu_45_01 = 0
mu_4_1 - mu_45_10 - mu_45_11 = 0
mu_4_0 - mu_47_00 - mu_47_01 = 0
mu_4_1 - mu_47_10 - mu_47_11 = 0
mu_5_0 - mu_25_00 - mu_25_10 = 0
mu_5_1 - mu_25_01 - mu_25_11 = 0
mu_5_0 - mu_45_00 - mu_45_10 = 0
mu_5_1 - mu_45_01 - mu_45_11 = 0
mu_5_0 - mu_58_00 - mu_58_01 = 0
mu_5_1 - mu_58_10 - mu_58_11 = 0
mu_6_0 - mu_36_00 - mu_36_10 = 0
mu_6_1 - mu_36_01 - mu_36_11 = 0
mu_6_0 - mu_67_00 - mu_67_01 = 0
mu_6_1 - mu_67_10 - mu_67_11 = 0
mu_7_0 - mu_47_00 - mu_47_10 = 0
mu_7_1 - mu_47_01 - mu_47_11 = 0
mu_7_0 - mu_67_00 - mu_67_10 = 0
mu_7_1 - mu_67_01 - mu_67_11 = 0
mu_7_0 - mu_78_00 - mu_78_01 = 0
mu_7_1 - mu_78_10 - mu_78_11 = 0
mu_8_0 - mu_58_00 - mu_58_10 = 0
mu_8_1 - mu_58_01 - mu_58_11 = 0
mu_8_0 - mu_78_00 - mu_78_10 = 0
mu_8_1 - mu_78_01 - mu_78_11 = 0
End)";

struct isotropic_dist_w_func {
    const int* primal_index;
    const int* num_bdds_var;
    double* dist_weights;
    const unsigned long num_vars;
    __device__ void operator()(const int i)
    {
        const int primal_var = primal_index[i];
        if (primal_var < num_vars) // ignores terminal nodes.
            dist_weights[i] = 1.0 / num_bdds_var[primal_var];
        else
            dist_weights[i] = 0.0;
    }
};

void test_problem(const char* instance, const double expected_lb, const double tol = 1e-8)
{
    ILP_input ilp = ILP_parser::parse_string(instance);
    bdd_preprocessor bdd_pre(ilp);
    bdd_collection bdd_col = bdd_pre.get_bdd_collection();
    bdd_cuda_learned_mma<double> solver(bdd_col);

    for(size_t i=0; i<solver.nr_variables(); ++i)
        solver.set_cost(ilp.objective()[i], i);

    std::vector<double> cost_vector_before = solver.get_primal_objective_vector_host();
    for(size_t i=0; i<solver.nr_variables(); ++i)
    {
        const auto diff = std::abs(ilp.objective()[i] - cost_vector_before[i]);
        std::stringstream buffer;
        buffer<<i<<" "<<ilp.objective()[i]<<" "<<cost_vector_before[i]<<" "<<diff<<"\n";
        test(diff <= tol, buffer.str());
    }

    const thrust::device_vector<int> primal_var_index = solver.get_primal_variable_index();
    const thrust::device_vector<int> num_bdds_var = solver.get_num_bdds_per_var();
    thrust::device_vector<double> dist_weights(primal_var_index.size());

    isotropic_dist_w_func func({thrust::raw_pointer_cast(primal_var_index.data()), 
                            thrust::raw_pointer_cast(num_bdds_var.data()), 
                            thrust::raw_pointer_cast(dist_weights.data()),
                            solver.nr_variables()});

    thrust::for_each(thrust::make_counting_iterator<int>(0), thrust::make_counting_iterator<int>(0) + dist_weights.size(), func);


    thrust::device_vector<double> sol_avg(primal_var_index.size());
    thrust::device_vector<double> lb_first_avg(solver.nr_bdds());
    thrust::device_vector<double> lb_second_avg(solver.nr_bdds());
    thrust::device_vector<double> direction_bfgs(sol_avg.size());

    solver.iterations(dist_weights.data(), 500, 0.5, 1e-9, sol_avg.data(), lb_first_avg.data(), lb_second_avg.data(), 20, 0.9, nullptr, 500, direction_bfgs.data());
    print_min_max(sol_avg.data(), "sol_avg", sol_avg.size());
    print_norm(direction_bfgs.data(), "bfgs", direction_bfgs.size());
    
    std::cout<<"Lower bound before distribute: "<<solver.lower_bound()<<", Expected: "<<expected_lb<<"\n";
    solver.distribute_delta();

    std::vector<double> cost_vector_after = solver.get_primal_objective_vector_host();
    for(size_t i=0; i<solver.nr_variables(); ++i)
    {
        const auto diff = std::abs(ilp.objective()[i] - cost_vector_after[i]);
        std::stringstream buffer;
        buffer<<i<<" "<<ilp.objective()[i]<<" "<<cost_vector_before[i]<<" "<<diff<<"\n";
        test(diff <= tol, buffer.str());
    }

    std::cout<<"Final lower bound: "<<solver.lower_bound()<<", Expected: "<<expected_lb<<"\n";
    test(std::abs(solver.lower_bound() - expected_lb) <= tol);
}

int main(int argc, char** argv)
{
    std::cout<<"two_simplex_non_unique_sols"<<"\n";
    test_problem(two_simplex_non_unique_sols, 3.0);
    std::cout<<"matching_3x3"<<"\n";
    test_problem(matching_3x3, -6.0);
    std::cout<<"short_chain_shuffled"<<"\n";
    test_problem(short_chain_shuffled, 1.0);
    std::cout<<"long_chain"<<"\n";
    test_problem(long_chain, -9.0);
    std::cout<<"grid_graph_3x3"<<"\n";
    test_problem(grid_graph_3x3, -8.0);
}

